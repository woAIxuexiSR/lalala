#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>
#include <tuple>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


template <class T = float>
class Image
{
public:
    int width, height, nchannel;
    T* data;

    Image(int _w, int _h, int _n = 3) : width(_w), height(_h), nchannel(_n)
    {
        checkCudaErrors(hipMallocManaged(&data, width * height * nchannel * sizeof(T)));
    }

    __host__ __device__ void set(int y, int x, T r, T g, T b)
    {
        int idx = y * width * nchannel + x * nchannel;
        data[idx] = r;
        data[idx + 1] = g;
        data[idx + 2] = b;
    }

    void save(const std::string& filename)
    {
        stbi_write_jpg(filename.c_str(), width, height, nchannel, data, 0);
    }

};


__global__ void kernel(int w, int h, Image<> img)
{
    // int ww = w / gridDim.x, hh = h / blockDim.x;

    // int x0 = blockIdx.x * ww, y0 = threadIdx.x * hh;
    // for(int i = 0; i < ww; i++)
    // {
    //     for(int j = 0; j < hh; j++)
    //     {
    //         int x = x0 + i, y = y0 + j;
    //         img.set(y, x, (float)x / w, (float)y / h, 0.0f);
    //     }
    // }

    for(int i = 0; i < w; i++)
    {
        for(int j = 0; j < h; j++)
        {
            // img.set(j, i, (float)i / w, (float)j / h, 0.0f);
            img.set(j, i, 1.0f, 0.0f, 0.0f);
        }
    }
}


int main()
{
    auto start = std::chrono::steady_clock::now();

    int w = 800, h = 600;
    Image<> img(w, h);
    kernel<<<1, 1>>>(w, h, img);
    hipDeviceSynchronize();

    img.save("test.jpg");

    auto end = std::chrono::steady_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::duration<double> >(end - start).count() << "s" << std::endl;
    return 0;
}